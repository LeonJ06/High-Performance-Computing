#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;

#include <math.h>
//#include <GL/glut.h>
//#include <GL/gl.h>
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include <algorithm>
#include "timer.h"

static int imagewidth;
static int imageheight;
static int pixellength;
static unsigned char* pixeldata;

static int imagewidth1;
static int imageheight1;
static int pixellength1;
static unsigned char* pixeldata1;



#define N 710*512    //图像分辨率
#define blocks 710   //圖像高度
#define threads 512  //圖像寬度





//CUDA  kernel函數********************************************************************

__global__ void add(int *a, int *r, int *g, int *b, float *gc)
{

	int i = (blockIdx.x*blockDim.x) + threadIdx.x;

//	for (int kk =0; kk < 100; kk++){	//increase computation
	gc[5120 * 6 + i * 6    ] = b[i] * 0.00390625;
	//gc[5120 * 6 + i * 6    ] = float(b[i]) / 256;
	gc[5120 * 6 + i * 6 + 1] = g[i] * 0.00390625;
	//gc[5120 * 6 + i * 6 + 1] = float(g[i]) / 256;
	gc[5120 * 6 + i * 6 + 2] = r[i] * 0.00390625;
	//gc[5120 * 6 + i * 6 + 2] = float(r[i]) / 256;

	gc[5120 * 6 + i * 6 + 3] = float(i - ((i>>9)<<9) );  // i%512
	//gc[5120 * 6 + i * 6 + 3] = float(i % 512);
	gc[5120 * 6 + i * 6 + 4] = float( i >> 9);
	//gc[5120 * 6 + i * 6 + 4] = float((i - (i % 512)) / 512);
	gc[5120 * 6 + i * 6 + 5] = float(a[i]);
//	}
}



float c[6 * N + 5120 * 6] = { 0.0 };
float f[6 * N + 5120 * 6] = { 0.0 };


int main(int argc, char* argv[]) {

	clock_t start, finish;
	//double totaltime;
	start = clock();
	//******************************************************************************

	//讀深度圖
	FILE* pfile = fopen("1.bmp", "rb");
	if (pfile == 0) exit(0);

	//讀取圖像大小
	fseek(pfile, 0x0012, SEEK_SET);
	fread(&imagewidth, sizeof(imagewidth), 1, pfile);
	fread(&imageheight, sizeof(imageheight), 1, pfile);

	//計算像素數據長度
	pixellength = imagewidth * 3;
	while (pixellength % 4 != 0)pixellength++;
	pixellength *= imageheight;

	//讀取像素數據
	pixeldata = (unsigned char*)malloc(pixellength);
	if (pixeldata == 0) exit(0);
	fseek(pfile, 54, SEEK_SET);
	//cout<<pixellength<<endl;
	fread(pixeldata, pixellength, 1, pfile);
	int shen[N];
	for (int i = 0; i <= N; i++)
		shen[i] = pixeldata[3 * i];

	//關閉文件
	fclose(pfile);

	//******************************************************************************

	//讀取亮度
	FILE* pfile1 = fopen("2.bmp", "rb");
	if (pfile1 == 0) exit(0);

	//讀取圖片大小
	fseek(pfile1, 0x0012, SEEK_SET);
	fread(&imagewidth1, sizeof(imagewidth1), 1, pfile1);
	fread(&imageheight1, sizeof(imageheight1), 1, pfile1);
	
	//計算數據長度
	pixellength1 = imagewidth1 * 3;
	while (pixellength1 % 4 != 0)pixellength1++;
	pixellength1 *= imageheight1;

	//讀取像素數據
	pixeldata1 = (unsigned char*)malloc(pixellength1);
	if (pixeldata1 == 0) exit(0);
	fseek(pfile1, 54, SEEK_SET);
	//cout<<pixellength<<endl;
	fread(pixeldata1, pixellength1, 1, pfile1);
	int red[N];
	int green[N];
	int blue[N];

	for (int i = 0; i <= N; i++)
	{
		red[i] = pixeldata1[3 * i];
		green[i] = pixeldata1[3 * i + 1];
		blue[i] = pixeldata1[3 * i + 2];
	}
	//關閉文件
	fclose(pfile1);

	//******************************************************************************

	//修補圖片
	int num = 0;
	for (int yo = 220; yo <= 390; yo++)//220,300
	{
		for (int xo = 212; xo <= 292; xo++)//212,292
		{
			if (shen[512 * yo + xo] == 0)	//如果深度=0  說明厝為了
			{

				for (int a = xo; a <= xo + 20; a++)
				{
					num++;					//一行一行找 紀錄缺失的總pixel
					if (shen[a + 512 * yo] != 0)
					{

						break;
					}
				}

				for (int r = 0; r<num; r++)	//對每一行 做線性修補
					shen[512 * yo + xo + r] = (shen[512 * yo + xo + r - 1] + shen[512 * yo + xo + r - 512]) / 2;

			}

		}
	}





	//******************************************************************************
	//深度圖缺失了
	int z1 = 0; int xbz = 0; int ybz = 0;
	for (int y0 = 220; y0 <= 390; y0++)//220,300
	{
		for (int x0 = 212; x0 <= 292; x0++)//212,292
		{
			if (shen[y0 * 512 + x0]>z1)//512
			{
				xbz = x0;
				ybz = y0;
				z1 = shen[y0 * 512 + x0];//512
			}
		}
	}

	int x1 = xbz - 90;//90
	int x2 = xbz + 90;
	int y1 = ybz - 90;
	int y2 = ybz + 90;
	cout << xbz << " " << 711 - ybz << endl;//513







	int s = 0; int n = 0, m = 0, j = 0, q = 0, k = 0;

	for (int y = y1; y <= y2; y++)
	{
		for (int x = x1; x <= x2; x++)
		{

			n = shen[y * 512 + x];//512
			m = shen[y * 512 + x + 1];
			j = blue[y * 512 + x + 1024];//b
			q = green[y * 512 + x + 1024];//g
			k = red[y * 512 + x + 1024];//r
			if (abs(n - m) >= 4)//4
								//if(abs(m-n)>=5&&abs(m-n)<=20)
			{
				for (int p = 1; p <= (abs(n - m) - 1); p++)
				{
					c[s * 6] = float(j) / 256;
					c[s * 6 + 1] = float(q) / 256;
					c[s * 6 + 2] = float(k) / 256;
					c[s * 6 + 3] = float(x);
					c[s * 6 + 4] = float(y);
					c[s * 6 + 5] = float(max(n, m) - p);


					s++;


				}
			}
		}
	}













	//CUDA計算部分*******************************************************************
	


        struct stopwatch_t* timerA1 = NULL;
	struct stopwatch_t* timerA2 = NULL;
	struct stopwatch_t* timerA3 = NULL;
        long double compA;
	long double commA1;
	long double commA2;
        /* initialize timer */
        stopwatch_init ();
        timerA1 = stopwatch_create ();
	timerA2 = stopwatch_create ();
	timerA3 = stopwatch_create ();


	int *dev_a = 0;
	int *dev_r = 0;
	int *dev_g = 0;
	int *dev_b = 0;
	float *dev_c = 0;

        int *dev_a2 = 0;
        int *dev_r2 = 0;
        int *dev_g2 = 0;
        int *dev_b2 = 0;
        float *dev_c2 = 0;

	int GPU_A = 0;
	int GPU_B = 1;

        //stopwatch_start (timerA1);
	hipSetDevice(GPU_A);

	hipMalloc((void**)&dev_a, (N>>1) * sizeof(int));
stopwatch_start (timerA1);
	hipMalloc((void**)&dev_r, (N>>1) * sizeof(int));
	hipMalloc((void**)&dev_g, (N>>1) * sizeof(int));
	hipMalloc((void**)&dev_b, (N>>1) * sizeof(int));
	hipMalloc((void**)&dev_c, 3 * (N + 5120) * sizeof(float));


	hipMemcpy(dev_a, shen, (N>>1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_r, red,  (N>>1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_g, green, (N>>1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, blue, (N>>1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, (N + 5120) * 3 * sizeof(float), hipMemcpyHostToDevice);

	commA1 = stopwatch_stop (timerA1);

	stopwatch_start (timerA2);
	add <<< blocks, threads >>>(dev_a, dev_r, dev_g, dev_b, dev_c);
	compA = stopwatch_stop (timerA2);

	stopwatch_start (timerA3);
	hipMemcpy(f, dev_c, 3 * (N + 5120) * sizeof(float), hipMemcpyDeviceToHost);	

	hipFree(dev_a);
	hipFree(dev_r);
	hipFree(dev_g);
	hipFree(dev_b);
	hipFree(dev_c);

	commA2 = stopwatch_stop (timerA3) + commA1;











        hipSetDevice(GPU_B);

        hipMalloc((void**)&dev_a2, (N>>1) * sizeof(int));
        hipMalloc((void**)&dev_r2, (N>>1) * sizeof(int));
        hipMalloc((void**)&dev_g2, (N>>1) * sizeof(int));
        hipMalloc((void**)&dev_b2, (N>>1) * sizeof(int));
        hipMalloc((void**)&dev_c2, 3 * (N + 5120) * sizeof(float));


        hipMemcpy(dev_a2, shen+(N>>1), (N>>1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_r2, red+(N>>1),  (N>>1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_g2, green+(N>>1), (N>>1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b2, blue+(N>>1), (N>>1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_c2, c+(N + 5120) * 3, (N + 5120) * 3 * sizeof(float), hipMemcpyHostToDevice);

        //dim3 grid(DIM,DIM);
        add <<< blocks, threads >>>(dev_a2, dev_r2, dev_g2, dev_b2, dev_c2);
        hipMemcpy(f+(N + 5120) * 3, dev_c2, 3 * (N + 5120) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(dev_a2);
        hipFree(dev_r2);
        hipFree(dev_g2);
        hipFree(dev_b2);
        hipFree(dev_c2);




        printf ("Computation time on GPU_A is: %Lg secs\n", compA);
	printf ("Communication time on GPU_A is: %Lg secs\n", commA2);






	return 0;


}

#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;
#include "timer.h"
#include <math.h>
//#include <GL/glut.h>
//#include <GL/gl.h>
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include <algorithm>

static int imagewidth;
static int imageheight;
static int pixellength;
static unsigned char* pixeldata;

static int imagewidth1;
static int imageheight1;
static int pixellength1;
static unsigned char* pixeldata1;



#define N 710*512    //图像分辨率
#define blocks 710   //圖像高度
#define threads 512  //圖像寬度

//CUDA  kernel********************************************************************

__global__ void add(int *a, int *r, int *g, int *b, float *gc)
{

	int i = (blockIdx.x*blockDim.x) + threadIdx.x;

	gc[5120 * 6 + i * 6    ] = b[i] * 0.00390625;
	//gc[5120 * 6 + i * 6    ] = float(b[i]) / 256;
	gc[5120 * 6 + i * 6 + 1] = g[i] * 0.00390625;
	//gc[5120 * 6 + i * 6 + 1] = float(g[i]) / 256;
	gc[5120 * 6 + i * 6 + 2] = r[i] * 0.00390625;
	//gc[5120 * 6 + i * 6 + 2] = float(r[i]) / 256;

	gc[5120 * 6 + i * 6 + 3] = float(i - ((i>>9)<<9) );  // i%512
	//gc[5120 * 6 + i * 6 + 3] = float(i % 512);
	gc[5120 * 6 + i * 6 + 4] = float( i >> 9);
	//gc[5120 * 6 + i * 6 + 4] = float((i - (i % 512)) / 512);
	gc[5120 * 6 + i * 6 + 5] = float(a[i]);
}



float c[6 * N + 5120 * 6] = { 0.0 };
float f[6 * N + 5120 * 6] = { 0.0 };

//////////////////////////
//			//
//			//
//	  Main		//
//			//
//			//
//////////////////////////
int main(int argc, char* argv[]) {

        struct stopwatch_t* timer = NULL;
	struct stopwatch_t* timer2 = NULL;
	struct stopwatch_t* timer3 = NULL;
        long double comp;
	long double comm;
	long double comm2;
        /* initialize timer */
        stopwatch_init ();
        timer = stopwatch_create ();
        timer2 = stopwatch_create ();
	timer3 = stopwatch_create ();
	//  Read 
	//******************************************************************************

	//讀深度圖
	FILE* pfile = fopen("1.bmp", "rb");
	if (pfile == 0) exit(0);

	//讀取圖像大小
	fseek(pfile, 0x0012, SEEK_SET);
	fread(&imagewidth, sizeof(imagewidth), 1, pfile);
	fread(&imageheight, sizeof(imageheight), 1, pfile);

	//計算像素數據長度
	pixellength = imagewidth * 3;
	while (pixellength % 4 != 0)pixellength++;
	pixellength *= imageheight;

	//讀取像素數據
	pixeldata = (unsigned char*)malloc(pixellength);
	if (pixeldata == 0) exit(0);
	fseek(pfile, 54, SEEK_SET);
	//cout<<pixellength<<endl;
	fread(pixeldata, pixellength, 1, pfile);
	int shen[N];
	for (int i = 0; i <= N; i++)
		shen[i] = pixeldata[3 * i];

	//關閉文件
	fclose(pfile);

	//******************************************************************************

	//讀取亮度
	FILE* pfile1 = fopen("2.bmp", "rb");
	if (pfile1 == 0) exit(0);

	//讀取圖片大小
	fseek(pfile1, 0x0012, SEEK_SET);
	fread(&imagewidth1, sizeof(imagewidth1), 1, pfile1);
	fread(&imageheight1, sizeof(imageheight1), 1, pfile1);
	
	//計算數據長度
	pixellength1 = imagewidth1 * 3;
	while (pixellength1 % 4 != 0)pixellength1++;
	pixellength1 *= imageheight1;

	//讀取像素數據
	pixeldata1 = (unsigned char*)malloc(pixellength1);
	if (pixeldata1 == 0) exit(0);
	fseek(pfile1, 54, SEEK_SET);
	//cout<<pixellength<<endl;
	fread(pixeldata1, pixellength1, 1, pfile1);
	int red[N];
	int green[N];
	int blue[N];

	for (int i = 0; i <= N; i++)
	{
		red[i] = pixeldata1[3 * i];
		green[i] = pixeldata1[3 * i + 1];
		blue[i] = pixeldata1[3 * i + 2];
	}

	fclose(pfile1);

	//****	Repair image	*******************************************************************

	
	int num = 0;
	for (int yo = 220; yo <= 390; yo++)//220,300
	{
		for (int xo = 212; xo <= 292; xo++)//212,292
		{
			if (shen[512 * yo + xo] == 0)	//如果深度=0  說明厝為了
			{

				for (int a = xo; a <= xo + 20; a++)
				{
					num++;					//一行一行找 紀錄缺失的總pixel
					if (shen[a + 512 * yo] != 0)
					{

						break;
					}
				}

				for (int r = 0; r<num; r++)	//對每一行 做線性修補
					shen[512 * yo + xo + r] = (shen[512 * yo + xo + r - 1] + shen[512 * yo + xo + r - 512]) / 2;

			}

		}
	}





	//******************************************************************************
	//深度圖缺失了
	int z1 = 0; int xbz = 0; int ybz = 0;
	for (int y0 = 220; y0 <= 390; y0++)//220,300
	{
		for (int x0 = 212; x0 <= 292; x0++)//212,292
		{
			if (shen[y0 * 512 + x0]>z1)//512
			{
				xbz = x0;
				ybz = y0;
				z1 = shen[y0 * 512 + x0];//512
			}
		}
	}

	int x1 = xbz - 90;//90
	int x2 = xbz + 90;
	int y1 = ybz - 90;
	int y2 = ybz + 90;
	cout << xbz << " " << 711 - ybz << endl;//513







	int s = 0; int n = 0, m = 0, j = 0, q = 0, k = 0;

	for (int y = y1; y <= y2; y++)
	{
		for (int x = x1; x <= x2; x++)
		{

			n = shen[y * 512 + x];//512
			m = shen[y * 512 + x + 1];
			j = blue[y * 512 + x + 1024];//b
			q = green[y * 512 + x + 1024];//g
			k = red[y * 512 + x + 1024];//r
			if (abs(n - m) >= 4)//4
								//if(abs(m-n)>=5&&abs(m-n)<=20)
			{
				for (int p = 1; p <= (abs(n - m) - 1); p++)
				{
					c[s * 6] = float(j) / 256;
					c[s * 6 + 1] = float(q) / 256;
					c[s * 6 + 2] = float(k) / 256;
					c[s * 6 + 3] = float(x);
					c[s * 6 + 4] = float(y);
					c[s * 6 + 5] = float(max(n, m) - p);


					s++;


				}
			}
		}
	}







	int *dev_a = 0;
	int *dev_r = 0;
	int *dev_g = 0;
	int *dev_b = 0;
	float *dev_c = 0;
	

	// assigned memory for arrays //
	hipMalloc((void**)&dev_a, N * sizeof(int));
stopwatch_start (timer2);
	hipMalloc((void**)&dev_r, N * sizeof(int));
	hipMalloc((void**)&dev_g, N * sizeof(int));

	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, 6 * (N + 5120) * sizeof(float));


	//  transfer data from CPU to GPU
	hipMemcpy(dev_a, shen, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_r, red, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_g, green, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, blue, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, (N + 5120) * 6 * sizeof(float), hipMemcpyHostToDevice);

comm = stopwatch_stop (timer2);

	//  start timer //
	stopwatch_start (timer);
	// execute kernel
	add <<< blocks, threads >>>(dev_a, dev_r, dev_g, dev_b, dev_c);
	//  stop the timer
	comp = stopwatch_stop (timer);



	stopwatch_start (timer3);
	// copy result to array f
	hipMemcpy(f, dev_c, 6 * (N + 5120) * sizeof(float), hipMemcpyDeviceToHost);


	hipFree(dev_a);
	hipFree(dev_r);
	hipFree(dev_g);
	hipFree(dev_b);
	hipFree(dev_c);
	
	
	comm += stopwatch_stop (timer3);

        printf ("Computation time on GPU is : %Lg secs\n", comp);
	printf ("Communication time on GPU and CPU is : %Lg secs\n", comm);

	return 0;


}

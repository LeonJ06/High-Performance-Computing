#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;


#include <gl/glut.h>

#define FileName "1.bmp"
static GLint imagewidth;
static GLint imageheight;
static GLint pixellength;
static GLubyte* pixeldata;
#include<stdio.h>
#include<stdlib.h>

int hui[4096];

int BMP()
{

 FILE* pfile=fopen("1.bmp","rb");
 if(pfile == 0) exit(0);

 fseek(pfile,0x0012,SEEK_SET);
 fread(&imagewidth,sizeof(imagewidth),1,pfile);
 fread(&imageheight,sizeof(imageheight),1,pfile);

 pixellength=imagewidth*3;
 while(pixellength%4 != 0)pixellength++;
 pixellength *= imageheight;

 pixeldata = (GLubyte*)malloc(pixellength);
 if(pixeldata == 0) exit(0);
 fseek(pfile,1078,SEEK_SET);
 cout<<pixellength<<endl;
 fread(pixeldata,pixellength,1,pfile);
  
for(int i=0;i<=4095;i++)
	hui[i]=pixeldata[i];
for(int j=3900;j<=4095;j++)
	cout<<hui[j]<<endl;



 fclose(pfile);

 return 0;
}




#define N 256

		__global__ void add(int *a,int *b)
		{int i=threadIdx.x;
		if(i<N)
			b[i*3]=i%64;
			b[i*3+1]=i/64;
			b[i*3+2]=a[i];
		}

	int main(){
		int a[N],b[3*N];
		int *dev_a=0;
		int *dev_b=0;
		for(int i=0;i<N;i++)
				{
					a[i]=hui[i];

					
				}
		hipMalloc((void**)&dev_a,N*sizeof(int));
		hipMalloc((void**)&dev_b,3*N*sizeof(int));


		

		hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);


		add<<<1,N>>>(dev_a,dev_b);

		hipMemcpy(b,dev_b,3*N*sizeof(int),hipMemcpyDeviceToHost);

		//for(int j=0;j<3*N;j++)
		//	cout<<b[j]<<endl;
		hipFree(dev_a);
		hipFree(dev_b);


		return 0;}


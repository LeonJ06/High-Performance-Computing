#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
using namespace std;

#include <math.h>
#include <GL/glut.h>
#include <GL/gl.h>
#include<stdio.h>
#include<stdlib.h>
#include <time.h>
#include <algorithm>

static GLint imagewidth;
static GLint imageheight;
static GLint pixellength;
static GLubyte* pixeldata;

static GLint imagewidth1;
static GLint imageheight1;
static GLint pixellength1;
static GLubyte* pixeldata1;



#define N 710*512    
#define blocks 710   
#define threads 512  

//CUDA  kernel********************************************************************

__global__ void add(int *a, int *r, int *g, int *b, float *gc)
{

	int i = (blockIdx.x*blockDim.x) + threadIdx.x;

	gc[5120 * 6 + i * 6    ] = b[i] * 0.00390625;
	//gc[5120 * 6 + i * 6    ] = float(b[i]) / 256;
	gc[5120 * 6 + i * 6 + 1] = g[i] * 0.00390625;
	//gc[5120 * 6 + i * 6 + 1] = float(g[i]) / 256;
	gc[5120 * 6 + i * 6 + 2] = r[i] * 0.00390625;
	//gc[5120 * 6 + i * 6 + 2] = float(r[i]) / 256;

	gc[5120 * 6 + i * 6 + 3] = float(i - ((i>>9)<<9) );  // i%512
	//gc[5120 * 6 + i * 6 + 3] = float(i % 512);
	gc[5120 * 6 + i * 6 + 4] = float( i >> 9);
	//gc[5120 * 6 + i * 6 + 4] = float((i - (i % 512)) / 512);
	gc[5120 * 6 + i * 6 + 5] = float(a[i]);
}



float c[6 * N + 5120 * 6] = { 0.0 };
float f[6 * N + 5120 * 6] = { 0.0 };



//openGL******************************************************************


GLint SCREEN_WIDTH = 0;
GLint SCREEN_HEIGHT = 0;

GLint windowWidth = 700;
GLint windowHeight = 700;

GLfloat xRotAngle = 0.0f;

GLfloat yRotAngle = 0.0f;

GLfloat zRotAngle = 0.0f;



void renderScreen(void) {


	//glClearColor(1.0f,1.0f,1.0f,1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LEQUAL);
	glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);

	glPushMatrix();

	glRotatef(xRotAngle, 1.0f, 0.0f, 0.0f);

	glRotatef(yRotAngle, 0.0f, 1.0f, 0.0f);

	glRotatef(zRotAngle, 0.0f, 0.0f, 1.0f);


	glEnable(GL_POINT_SMOOTH);
	glHint(GL_POINT_SMOOTH, GL_NICEST);
	glEnable(GL_LINE_SMOOTH);
	glHint(GL_LINE_SMOOTH, GL_NICEST);



	glColor3f(1.0f, 1.0f, 1.0f);
	glBegin(GL_LINES);
	glVertex3f(-53.0f, 0.0f, 0.0f);
	glVertex3f(53.0f, 0.0f, 0.0f);
	glVertex3f(0.0f, -53.0f, 0.0f);
	glVertex3f(0.0f, 53.0f, 0.0f);
	glVertex3f(0.0f, 0.0f, -53.0f);
	glVertex3f(0.0f, 0.0f, 53.0f);
	glEnd();

	glPushMatrix();
	glTranslatef(53.0f, 0.0f, 0.0f);
	glRotatef(90.0f, 0.0f, 1.0f, 0.0f);
	glutWireCone(10, 20, 10, 10);
	glPopMatrix();

	glPushMatrix();
	glTranslatef(0.0f, 53.0f, 0.0f);
	glRotatef(-90.0f, 1.0f, 0.0f, 0.0f);
	glutWireCone(10, 20, 10, 10);
	glPopMatrix();

	glPushMatrix();
	glTranslatef(0.0f, 0.0f, 53.0f);
	glRotatef(90.0f, 0.0f, 0.0f, 1.0f);
	glutWireCone(10, 20, 10, 10);
	glPopMatrix();




	glEnableClientState(GL_COLOR_ARRAY);
	glEnableClientState(GL_VERTEX_ARRAY);
	glColorPointer(3, GL_FLOAT, 6 * sizeof(GLfloat), &f[0]);
	glVertexPointer(3, GL_FLOAT, 6 * sizeof(GLfloat), &f[3]);
	glPointSize(1);
	glBegin(GL_POINTS);
	for (int i = 0; i<(N + 5120); i++)

		glArrayElement(i);


	glEnd();

	glPopMatrix();

	glutSwapBuffers();
}


void changeSize(GLint w, GLint h) {

	GLfloat ratio;

	GLfloat coordinatesize = 750.0f;

	if ((w == 0) || (h == 0))
		return;

	glViewport(0, 0, w * 1, h * 1);
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	ratio = (GLfloat)w / (GLfloat)h;



	glOrtho(-coordinatesize, coordinatesize, -coordinatesize / ratio, coordinatesize / ratio, -coordinatesize, coordinatesize);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
}


void specialKey(int key, int x, int y) {

	if (key == GLUT_KEY_UP) {
		xRotAngle -= 5.0f;
	}
	else if (key == GLUT_KEY_DOWN) {
		xRotAngle += 5.0f;
	}
	else if (key == GLUT_KEY_LEFT) {
		yRotAngle -= 5.0f;
	}
	else if (key == GLUT_KEY_RIGHT) {
		yRotAngle += 5.0f;
	}
	else if (key == GLUT_KEY_PAGE_UP) {
		zRotAngle -= 5.0f;
	}
	else if (key == GLUT_KEY_PAGE_DOWN) {
		zRotAngle += 5.0f;
	}

	glutPostRedisplay();
}





int main(int argc, char* argv[]) {

	clock_t start, finish;
	double totaltime;
	start = clock();
	//******************************************************************************

	FILE* pfile = fopen("1.bmp", "rb");
	if (pfile == 0) exit(0);


	fseek(pfile, 0x0012, SEEK_SET);
	fread(&imagewidth, sizeof(imagewidth), 1, pfile);
	fread(&imageheight, sizeof(imageheight), 1, pfile);


	pixellength = imagewidth * 3;
	while (pixellength % 4 != 0)pixellength++;
	pixellength *= imageheight;


	pixeldata = (GLubyte*)malloc(pixellength);
	if (pixeldata == 0) exit(0);
	fseek(pfile, 54, SEEK_SET);
	//cout<<pixellength<<endl;
	fread(pixeldata, pixellength, 1, pfile);
	int shen[N];
	for (int i = 0; i <= N; i++)
		shen[i] = pixeldata[3 * i];


	fclose(pfile);

	//******************************************************************************

	FILE* pfile1 = fopen("2.bmp", "rb");
	if (pfile1 == 0) exit(0);


	fseek(pfile1, 0x0012, SEEK_SET);
	fread(&imagewidth1, sizeof(imagewidth1), 1, pfile1);
	fread(&imageheight1, sizeof(imageheight1), 1, pfile1);
	

	pixellength1 = imagewidth1 * 3;
	while (pixellength1 % 4 != 0)pixellength1++;
	pixellength1 *= imageheight1;


	pixeldata1 = (GLubyte*)malloc(pixellength1);
	if (pixeldata1 == 0) exit(0);
	fseek(pfile1, 54, SEEK_SET);
	//cout<<pixellength<<endl;
	fread(pixeldata1, pixellength1, 1, pfile1);
	int red[N];
	int green[N];
	int blue[N];

	for (int i = 0; i <= N; i++)
	{
		red[i] = pixeldata1[3 * i];
		green[i] = pixeldata1[3 * i + 1];
		blue[i] = pixeldata1[3 * i + 2];
	}

	fclose(pfile1);

	//******************************************************************************


	/*		 c[0]=100;
	c[1]=100;
	c[2]=100;
	c[3]=-10;
	c[4]=-10;
	c[5]=-10;
	*/

	//******************************************************************************

	//修補圖片
	int num = 0;
	for (int yo = 220; yo <= 390; yo++)//220,300
	{
		for (int xo = 212; xo <= 292; xo++)//212,292
		{
			if (shen[512 * yo + xo] == 0)	//如果深度=0  說明厝為了
			{

				for (int a = xo; a <= xo + 20; a++)
				{
					num++;					//一行一行找 紀錄缺失的總pixel
					if (shen[a + 512 * yo] != 0)
					{

						break;
					}
				}

				for (int r = 0; r<num; r++)	//對每一行 做線性修補
					shen[512 * yo + xo + r] = (shen[512 * yo + xo + r - 1] + shen[512 * yo + xo + r - 512]) / 2;

			}

		}
	}





	//******************************************************************************
	//深度圖缺失了
	int z1 = 0; int xbz = 0; int ybz = 0;
	for (int y0 = 220; y0 <= 390; y0++)//220,300
	{
		for (int x0 = 212; x0 <= 292; x0++)//212,292
		{
			if (shen[y0 * 512 + x0]>z1)//512
			{
				xbz = x0;
				ybz = y0;
				z1 = shen[y0 * 512 + x0];//512
			}
		}
	}

	int x1 = xbz - 90;//90
	int x2 = xbz + 90;
	int y1 = ybz - 90;
	int y2 = ybz + 90;
	cout << xbz << " " << 711 - ybz << endl;//513







	int s = 0; int n = 0, m = 0, j = 0, q = 0, k = 0;

	for (int y = y1; y <= y2; y++)
	{
		for (int x = x1; x <= x2; x++)
		{

			n = shen[y * 512 + x];//512
			m = shen[y * 512 + x + 1];
			j = blue[y * 512 + x + 1024];//b
			q = green[y * 512 + x + 1024];//g
			k = red[y * 512 + x + 1024];//r
			if (abs(n - m) >= 4)//4
								//if(abs(m-n)>=5&&abs(m-n)<=20)
			{
				for (int p = 1; p <= (abs(n - m) - 1); p++)
				{
					c[s * 6] = float(j) / 256;
					c[s * 6 + 1] = float(q) / 256;
					c[s * 6 + 2] = float(k) / 256;
					c[s * 6 + 3] = float(x);
					c[s * 6 + 4] = float(y);
					c[s * 6 + 5] = float(max(n, m) - p);


					s++;


				}
			}
		}
	}













	//CUDA計算部分*******************************************************************


	int *dev_a = 0;
	int *dev_r = 0;
	int *dev_g = 0;
	int *dev_b = 0;
	float *dev_c = 0;



	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_r, N * sizeof(int));
	hipMalloc((void**)&dev_g, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, 6 * (N + 5120) * sizeof(float));


	hipMemcpy(dev_a, shen, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_r, red, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_g, green, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, blue, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, (N + 5120) * 6 * sizeof(float), hipMemcpyHostToDevice);

	float time_elapsed=0;
	hipEvent_t start1;
	hipEvent_t stop;
	
	hipEventCreate(&start1);    //創建EVENT
	hipEventCreate(&stop);

	hipEventRecord( start1,0);    //紀錄當前時間
	

	//dim3 grid(DIM,DIM);
	add <<< blocks, threads >>>(dev_a, dev_r, dev_g, dev_b, dev_c);

			hipEventRecord( stop,0);    //紀錄當前時間

	hipEventSynchronize(start1);    //Waits for an event to complete.
	hipEventSynchronize(stop);    //Waits for an event to complete.Record之前任務
	hipEventElapsedTime(&time_elapsed,start1,stop);    //計算時間差
	



	hipMemcpy(f, dev_c, 6 * (N + 5120) * sizeof(float), hipMemcpyDeviceToHost);

	//for(int j=24500;j<24576;j++)
	//	cout<<c[j]<<endl;

	hipFree(dev_a);
	hipFree(dev_r);
	hipFree(dev_g);
	hipFree(dev_b);
	hipFree(dev_c);

	//hipEventDestroy(start);    //destory the event
	hipEventDestroy(stop);

	printf("執行時間%f(ms)\n",time_elapsed);		//cuda執行時間
	

	//openGL顯示部分*****************************************************************



	glutInit(&argc, argv);

	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);

	SCREEN_WIDTH = glutGet(GLUT_SCREEN_WIDTH);

	SCREEN_HEIGHT = glutGet(GLUT_SCREEN_HEIGHT);

	glutCreateWindow("3D HumanFace With CUDA&openGL");

	glutReshapeWindow(windowWidth, windowHeight);

	glutPositionWindow((SCREEN_WIDTH - windowWidth) / 2, (SCREEN_HEIGHT - windowHeight) / 2);

	glutReshapeFunc(changeSize);

	glutDisplayFunc(renderScreen);

	glutSpecialFunc(specialKey);

	finish = clock();
	totaltime = (double)(finish - start) / CLOCKS_PER_SEC;
	cout<<"\n此程序運行時間為"<<totaltime*1000<<"毫秒"<<endl;

	glutMainLoop();



	return 0;


}
